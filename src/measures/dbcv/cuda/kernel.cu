
#include <hip/hip_runtime.h>
 /**
  * Given two objects in a dataset, finds the euclidean distance between them.
  * params:
  * i_a: index of first object;
  * i_b: index of second object;
  * n_attributes: dimensionality of dataset;
  * dataset: dataset store in an unidimensional array of shape n_objects * n_attributes;
  */
 __device__ float euclidean_distance(int i_a, int i_b, int n_attributes, float *dataset) {
    int j;
    float distance = 0;
    for(j = 0; j < n_attributes; j++) {
        distance += powf(
            dataset[(i_a * n_attributes) + j] - dataset[(i_b * n_attributes) + j],
            2
        );
    }
    return distance;
}

 __global__ void distance_matrix(float *distance_matrix, int n_objects, int n_attributes, float *dataset) {
	const int idx = threadIdx.x + (blockDim.x * blockIdx.x);
	if(idx < n_objects) {
		int i;
		for(i = 0; i < n_objects; i++) {
			distance_matrix[(idx * n_objects) + i] = euclidean_distance(i, idx, n_attributes, dataset);
		}
	}
 }

 __global__ void all_pts_coredist(float *coredist, int n_objects, int n_attributes, float *dataset, int *labels) {

     const int idx = threadIdx.x + blockDim.x * blockIdx.x;

     if(idx < n_objects) {
        int i;

        float numeral = 0;
        int objects_in_cluster = 0;

        for(i = 0; i < n_objects; i++) {
            int within_group = (labels[idx] == labels[i]);
            objects_in_cluster += within_group;

            float distance = euclidean_distance(idx, i, n_attributes, dataset);

            float temp = powf(
                ((float)1 / distance),
                n_attributes
            );
            numeral += (float)within_group * (temp * (float)(idx != i));
        }

        float core_dist = powf(
            (objects_in_cluster - 1) / numeral,
            ((float)1/n_attributes)
        );

        coredist[idx] = core_dist;
     }
}

__global__ void global_mreach_matrix(float *distance_matrix, float *mreach, float *coredist, int n_objects) {
	const int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if(idx < n_objects) {
		int i;
		float mreach_dist, distance;
		for(i = 0; i < n_objects; i++) {
			distance = distance_matrix[(idx * n_objects) + i];  // euclidean distance from objects i and idx
			mreach_dist = fmaxf(
				fmaxf(
					coredist[idx],
					coredist[i]
				),
				distance
			);
			mreach[(idx * n_objects) + i] = mreach_dist;
		}
	}
}

 __global__ void mreach_matrix(float *distance_matrix, float *mreach, float *coredist, int cluster, int *labels, int n_objects, int n_attributes, float *dataset) {
	const int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if(idx < n_objects) {
		int i;
		float mreach_dist, distance;
		for(i = 0; i < n_objects; i++) {
			distance = distance_matrix[(idx * n_objects) + i];
			mreach_dist = fmaxf(
				fmaxf(
					coredist[idx],
					coredist[i]
				),
				distance
			);
			mreach[(idx * n_objects) + i] = ((labels[idx] == cluster) && (labels[i] == cluster)) * mreach_dist;
				// + (((labels[idx] != cluster) || (labels[i] != cluster)) * 0);  // 0 denotes no link between objects
		}
	}
 }